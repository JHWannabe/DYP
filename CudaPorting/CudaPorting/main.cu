#include "hip/hip_runtime.h"
#include "Eigen/Core"
#include "Eigen/Dense"
#include "opencv2/opencv.hpp"
#include "opencv2/core/eigen.hpp"
#include "opencv2/highgui/highgui.hpp"
#include <iostream>
#include <vector>
#include <ctime>

#include "hip/hip_runtime.h"
#include ""

using namespace cv;
using namespace Eigen;
using namespace std;

#define M_IMAGE_COUNT 4

Eigen::MatrixXd LightMatrixPinv(const std::vector<Eigen::Vector4f>& lightMat) {
    int numRows = lightMat.size();
    int numCols = (numRows > 0) ? lightMat[0].size() : 0;

    Eigen::MatrixXd matrix(numRows, numCols);
    for (int i = 0; i < numRows; ++i) {
        matrix.row(i) = lightMat[i].cast<double>();
    }

    Eigen::JacobiSVD<Eigen::MatrixXd> svd(matrix, Eigen::ComputeThinU | Eigen::ComputeThinV);
    Eigen::MatrixXd singularValuesInv = svd.singularValues().unaryExpr([&](double sv) {
        return (sv > 1e-8) ? (1.0 / sv) : 0.0;
        });
    return svd.matrixV() * singularValuesInv.asDiagonal() * svd.matrixU().transpose();
}

__global__ void normalizeImage(double* image, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        image[idx] /= 255.0;
    }
}

__global__ void matrixMultiplyKernel(const double* a, const double* b, double* c, int m, int n, int l) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < m && col < l) {
        double sum = 0.0;
        for (int k = 0; k < n; ++k) {
            double a_val = a[row + m * k];
            double b_val = b[col * n + k];
            sum += a_val * b_val;
        }
        c[row + m * col] = sum;
    }
}

__global__ void rowNormKernel(const double* matrixData, double* normsData, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        double l2_norm_squared = 0.0;
        for (int col = 0; col < cols; ++col) {
            double element = matrixData[row * cols + col];
            l2_norm_squared += element * element;
        }
        normsData[row] = sqrt(l2_norm_squared);
    }
}

__global__ void clipKernel(const double* input, double* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        double value = input[idx];
        output[idx] = fmin(fmax(value, 0.0), 1.0);
    }
}

__global__ void reshapeMatrixKernel(const double* vec, double* matrix, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        matrix[row * cols + col] = vec[row * cols + col];
    }
}

__global__ void flipKernel(const double* albedo, uchar* result, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < cols && idy < rows) {
        int index = idy * cols + idx;
        result[index] = static_cast<uchar>(albedo[index] * 255.0f);
    }
}

__global__ void elementWiseDivisionKernel(const double* inputMatrix, const double* normVector, double* outputMatrix, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < rows && j < cols) {
        int index = i + rows * j;
        outputMatrix[index] = inputMatrix[index] / normVector[i];
    }
}

__global__ void setZerosToOnes(double* _rhoData, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < rows) {
        if (_rhoData[i * cols + 2] == 0)
            _rhoData[i * cols + 2] = 1;
    }
}

__global__ void transposeMatrixKernel(const double* inputMatrix, double* outputMatrix, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < rows && col < cols) {
        if (col == 0) {
            outputMatrix[row] = inputMatrix[row];
        }
    }
}

__global__ void copyKernel(double* A, double* B, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < rows && j < cols) {
        B[i + rows * j] = A[i * cols + j] / 255.0;
    }
}


int main() {
    int threadsPerBlock1 = 1024;
    dim3 threadsPerBlock2(32, 32);

    cv::Mat img0, img1, img2, img3;
    time_t _tstart, _tend, mid;
    _tstart = clock();

    img0 = cv::imread("resize/0_B0.bmp", cv::IMREAD_GRAYSCALE);
    img1 = cv::imread("resize/0_B1.bmp", cv::IMREAD_GRAYSCALE);
    img2 = cv::imread("resize/0_B2.bmp", cv::IMREAD_GRAYSCALE);
    img3 = cv::imread("resize/0_B3.bmp", cv::IMREAD_GRAYSCALE);

    int _rows = img0.rows;
    int _cols = img0.cols;
    int _size = _rows * _cols;

    std::vector<Eigen::Vector4f> lightMat;
    lightMat.emplace_back(-0.6133723, -0.6133723, 0.6133723, 0.6133723);
    lightMat.emplace_back(-0.613372, 0.613372, 0.613372, -0.613372);
    lightMat.emplace_back(0.49754286, 0.49754286, 0.49754286, 0.49754286);

    Eigen::MatrixXd _lightMatpinv = LightMatrixPinv(lightMat);

    std::vector<double> image0Data(img0.ptr<uchar>(), img0.ptr<uchar>() + _size);
    std::vector<double> image1Data(img1.ptr<uchar>(), img1.ptr<uchar>() + _size);
    std::vector<double> image2Data(img2.ptr<uchar>(), img2.ptr<uchar>() + _size);
    std::vector<double> image3Data(img3.ptr<uchar>(), img3.ptr<uchar>() + _size);

    Eigen::MatrixXd _merged_matrix(_size, 4);
    _merged_matrix.col(0) = Eigen::VectorXd::Map(image0Data.data(), _size);
    _merged_matrix.col(1) = Eigen::VectorXd::Map(image1Data.data(), _size);
    _merged_matrix.col(2) = Eigen::VectorXd::Map(image2Data.data(), _size);
    _merged_matrix.col(3) = Eigen::VectorXd::Map(image3Data.data(), _size);

    Eigen::MatrixXd _rho_t(_size, 3);
    cv::Mat cvMatResult(_rows, _cols, CV_8UC1);
    double* d_lightMatpinv, * d_rho_t = new double[_size * 3];
    double* d_matrix, * d_norm, * d_norm_t;
    double* d_rho, * d_transposed_rho, * d_n;
    uchar* d_result;
    double* d_merged_matrix;

    hipMalloc((void**)&d_merged_matrix, 4 * _size * sizeof(double));
    hipMalloc((void**)&d_lightMatpinv, 4 * _size * sizeof(double));
    hipMalloc((void**)&d_rho_t, 3 * _size * sizeof(double));
    hipMalloc((void**)&d_matrix, _size * sizeof(double));
    hipMalloc((void**)&d_norm, _size * sizeof(double));
    hipMalloc((void**)&d_norm_t, _size * sizeof(double));
    hipMalloc((void**)&d_result, sizeof(uchar) * _rows * _cols);
    hipMalloc((void**)&d_rho, 3 * _size * sizeof(double));
    hipMalloc((void**)&d_transposed_rho, 3 * _size * sizeof(double));
    hipMalloc((void**)&d_n, _size * sizeof(double));

    hipMemcpy(d_merged_matrix, _merged_matrix.data(), 4 * _size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_lightMatpinv, _lightMatpinv.data(), 4 * 3 * sizeof(double), hipMemcpyHostToDevice);

    int blocksPerGrid1 = (4 * _size + threadsPerBlock1 - 1) / threadsPerBlock1;
    dim3 blocksPerGrid2((_size + threadsPerBlock2.x - 1) / threadsPerBlock2.x, (3 + threadsPerBlock2.y - 1) / threadsPerBlock2.y);
    dim3 blocksPerGrid3((_rows + threadsPerBlock2.x - 1) / threadsPerBlock2.x, (_cols + threadsPerBlock2.y - 1) / threadsPerBlock2.y);
    dim3 blocksPerGrid4((_cols + threadsPerBlock2.x - 1) / threadsPerBlock2.x, (_rows + threadsPerBlock2.y - 1) / threadsPerBlock2.y);

    normalizeImage << <blocksPerGrid1, threadsPerBlock1 >> > (d_merged_matrix, 4 * _size);

    blocksPerGrid1 = (_size + threadsPerBlock1 - 1) / threadsPerBlock1;

    matrixMultiplyKernel << <blocksPerGrid2, threadsPerBlock2 >> > (d_merged_matrix, d_lightMatpinv, d_rho_t, _size, 4, 3);
    rowNormKernel << <blocksPerGrid1, threadsPerBlock1 >> > (&d_rho_t[2 * _size], d_norm, _size, 1);
    clipKernel << <blocksPerGrid1, threadsPerBlock1 >> > (d_norm, d_norm_t, _size);
    reshapeMatrixKernel << <blocksPerGrid3, threadsPerBlock2 >> > (d_norm_t, d_matrix, _cols, _rows);
    flipKernel << <blocksPerGrid4, threadsPerBlock2 >> > (d_matrix, d_result, _rows, _cols);

    blocksPerGrid1 = ((3 + threadsPerBlock1 - 1) / threadsPerBlock1);

    elementWiseDivisionKernel << <blocksPerGrid2, threadsPerBlock2 >> > (d_rho_t, d_norm_t, d_rho, _size, 3);
    setZerosToOnes << <blocksPerGrid1, threadsPerBlock1 >> > (d_rho, _size, 3);
    transposeMatrixKernel << <blocksPerGrid2, threadsPerBlock2 >> > (d_rho, d_transposed_rho, _size, 3);
    copyKernel << <blocksPerGrid3, threadsPerBlock2 >> > (d_transposed_rho, d_n, _rows, _cols);

    Eigen::MatrixXd col0(_rows, _cols);

    hipMemcpy(cvMatResult.data, d_result, sizeof(uchar) * _rows * _cols, hipMemcpyDeviceToHost);
    hipMemcpy(col0.data(), d_n, _size * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_lightMatpinv);
    hipFree(d_merged_matrix);
    hipFree(d_norm);
    hipFree(d_matrix);
    hipFree(d_result);
    hipFree(d_rho_t);
    hipFree(d_norm_t);
    hipFree(d_rho);
    hipFree(d_n);
    hipFree(d_transposed_rho);

    cv::Mat _normalmap, normalmap_cv(_rows, _cols, CV_64FC1);

    for (int i = 0; i < _rows; ++i) {
        for (int j = 0; j < _cols; ++j) {
            normalmap_cv.at<double>(i, j) = col0(i, j);
        }
    }

    cv::normalize(cvMatResult, cvMatResult, 0, 255, cv::NORM_MINMAX, CV_8UC1);
    cv::normalize(normalmap_cv, _normalmap, 0, 255, cv::NORM_MINMAX, CV_8UC3);

    _tend = clock();
    cout << "����ð� : " << (float)(_tend - _tstart) / 1000 << " s" << endl;

    cv::imwrite("result/0_albedo0.bmp", cvMatResult);
    cv::imwrite("result/0_albedo1.bmp", _normalmap);

    waitKey(2000);

}