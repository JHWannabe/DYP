#include "hip/hip_runtime.h"
#include "Eigen/Core"
#include "Eigen/Dense"
#include "opencv2/opencv.hpp"
#include "opencv2/core/eigen.hpp"
#include "opencv2/highgui/highgui.hpp"
#include <iostream>
#include <vector>
#include <ctime>

#include "hip/hip_runtime.h"
#include ""

using namespace cv;
using namespace Eigen;
using namespace std;

#define M_IMAGE_COUNT 4

Eigen::MatrixXd LightMatrixPinv(const std::vector<Eigen::Vector4f>& lightMat) {
    int numRows = lightMat.size();
    int numCols = (numRows > 0) ? lightMat[0].size() : 0;

    Eigen::MatrixXd matrix(numRows, numCols);
    for (int i = 0; i < numRows; ++i) {
        matrix.row(i) = lightMat[i].cast<double>();
    }

    Eigen::JacobiSVD<Eigen::MatrixXd> svd(matrix, Eigen::ComputeThinU | Eigen::ComputeThinV);
    Eigen::MatrixXd singularValuesInv = svd.singularValues().unaryExpr([&](double sv) {
        return (sv > 1e-8) ? (1.0 / sv) : 0.0;
        });
    return svd.matrixV() * singularValuesInv.asDiagonal() * svd.matrixU().transpose();
}

__global__ void copyMatrixToCUDA(const double* image0Data, const double* image1Data, const double* image2Data, const double* image3Data, double* d_merged_matrix, int _size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < _size && j < M_IMAGE_COUNT) {
        d_merged_matrix[i + _size * j] = (j == 0) ? image0Data[i] :
            (j == 1) ? image1Data[i] :
            (j == 2) ? image2Data[i] :
            image3Data[i];
    }
}

__global__ void normalizeImage(double* image, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        image[idx] /= 255.0;
    }
}

__global__ void matrixMultiplyKernel(const double* a, const double* b, double* c, int m, int n, int l) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < m && j < l) {
        double sum = 0.0;
        for (int k = 0; k < n; ++k) {
            double a_val = a[i + m * k];
            double b_val = b[j * n + k];
            sum += a_val * b_val;
        }
        c[i + m * j] = sum;
    }
}

__global__ void rowNormKernel(const double* matrixData, double* normsData, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < rows) {
        double l2_norm_squared = 0.0;
        for (int j = 0; j < cols; ++j) {
            double element = matrixData[i * cols + j];
            l2_norm_squared += element * element;
        }
        normsData[i] = sqrt(l2_norm_squared);
    }
}

__global__ void clipKernel(const double* input, double* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        double value = input[idx];
        output[idx] = fmin(fmax(value, 0.0), 1.0);
    }
}

__global__ void reshapeMatrixKernel(const double* vec, double* matrix, int rows, int cols) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < rows && j < cols) {
        matrix[i * cols + j] = vec[i * cols + j];
    }
}

__global__ void flipKernel(const double* albedo, uchar* result, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < cols && j < rows) {
        int index = i + cols * j;
        result[index] = static_cast<uchar>(albedo[index] * 255.0f);
    }
}

__global__ void elementWiseDivisionKernel(const double* inputMatrix, const double* normVector, double* outputMatrix, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < rows && j < cols) {
        int index = i + rows * j;
        outputMatrix[index] = inputMatrix[index] / normVector[i];
    }
}

__global__ void setZerosToOnes(double* _rhoData, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < rows) {
        if (_rhoData[i * cols + 2] == 0)
            _rhoData[i * cols + 2] = 1;
    }
}

__global__ void copyKernel(double* A, double* B, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < rows && j < cols) {
        B[i * cols + j] = A[i * cols + j] / 255.0;
    }
}

__global__ void minMaxKernel(double* A, int rows, int cols, double* maxVal, double* minVal) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < rows && j < cols) {
        *minVal = (A[i * cols + j] < *minVal) ? A[i * cols + j] : *minVal;
        *maxVal = (A[i * cols + j] > *maxVal) ? A[i * cols + j] : *maxVal;
    }
}


__global__ void normalizeKernel(const double* input, uchar3* output, int rows, int cols, double* maxVal, double* minVal) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < cols) {
        // Normalize the input values and convert to uchar3
        double normalizedValue = 255.0 * (input[i * cols + j] - *minVal) / (*maxVal - *minVal);
        uchar3 result;
        result.x = static_cast<uchar>(normalizedValue);
        result.y = static_cast<uchar>(normalizedValue);
        result.z = static_cast<uchar>(normalizedValue);
        output[i * cols + j] = result;
    }
}


int main() {
    int threadsPerBlock1 = 1024;
    dim3 threadsPerBlock2(32, 32);
    hipStream_t s1, s2, s3, s4;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);
    hipStreamCreate(&s3);
    hipStreamCreate(&s4);

    cv::Mat img0, img1, img2, img3;
    time_t _tstart, _tend;
    _tstart = clock();

    img0 = cv::imread("resize/0_B0.bmp", cv::IMREAD_GRAYSCALE);
    img1 = cv::imread("resize/0_B1.bmp", cv::IMREAD_GRAYSCALE);
    img2 = cv::imread("resize/0_B2.bmp", cv::IMREAD_GRAYSCALE);
    img3 = cv::imread("resize/0_B3.bmp", cv::IMREAD_GRAYSCALE);

    int _rows = img0.rows;
    int _cols = img0.cols;
    int _size = _rows * _cols;

    std::vector<Eigen::Vector4f> lightMat;
    lightMat.emplace_back(-0.6133723, -0.6133723, 0.6133723, 0.6133723);
    lightMat.emplace_back(-0.613372, 0.613372, 0.613372, -0.613372);
    lightMat.emplace_back(0.49754286, 0.49754286, 0.49754286, 0.49754286);

    Eigen::MatrixXd _lightMatpinv = LightMatrixPinv(lightMat);

    std::vector<double> image0Data(img0.ptr<uchar>(), img0.ptr<uchar>() + _size);
    std::vector<double> image1Data(img1.ptr<uchar>(), img1.ptr<uchar>() + _size);
    std::vector<double> image2Data(img2.ptr<uchar>(), img2.ptr<uchar>() + _size);
    std::vector<double> image3Data(img3.ptr<uchar>(), img3.ptr<uchar>() + _size);

    Eigen::MatrixXd _merged_matrix(_size, M_IMAGE_COUNT);

    double minVal = 1e9, maxVal = -1e9;
    double* d_merged_matrix, * d_image0Data, * d_image1Data, * d_image2Data, * d_image3Data;
    double* d_lightMatpinv, * d_rho_t = new double[_size * 3];
    double* d_matrix, * d_norm, * d_norm_t, * d_rho, * d_transposed_rho, * d_n, * d_min, * d_max;
    uchar* d_result;
    uchar3* d_output;

    hipMalloc((void**)&d_merged_matrix, M_IMAGE_COUNT * _size * sizeof(double));
    hipMalloc((void**)&d_image0Data, _size * sizeof(double));
    hipMalloc((void**)&d_image1Data, _size * sizeof(double));
    hipMalloc((void**)&d_image2Data, _size * sizeof(double));
    hipMalloc((void**)&d_image3Data, _size * sizeof(double));
    hipMalloc((void**)&d_lightMatpinv, M_IMAGE_COUNT * _size * sizeof(double));
    hipMalloc((void**)&d_rho_t, 3 * _size * sizeof(double));
    hipMalloc((void**)&d_matrix, _size * sizeof(double));
    hipMalloc((void**)&d_norm, _size * sizeof(double));
    hipMalloc((void**)&d_norm_t, _size * sizeof(double));
    hipMalloc((void**)&d_result, _size * sizeof(uchar));
    hipMalloc((void**)&d_rho, 3 * _size * sizeof(double));
    hipMalloc((void**)&d_n, _size * sizeof(double));
    hipMalloc((void**)&d_min, sizeof(double));
    hipMalloc((void**)&d_max, sizeof(double));
    hipMalloc((void**)&d_output, _rows * _cols * sizeof(uchar3));

    // Copy image data to device memory
    hipMemcpyAsync(d_image0Data, image0Data.data(), _size * sizeof(double), hipMemcpyHostToDevice, s1);
    hipMemcpyAsync(d_image1Data, image1Data.data(), _size * sizeof(double), hipMemcpyHostToDevice, s2);
    hipMemcpyAsync(d_image2Data, image2Data.data(), _size * sizeof(double), hipMemcpyHostToDevice, s3);
    hipMemcpyAsync(d_image3Data, image3Data.data(), _size * sizeof(double), hipMemcpyHostToDevice, s4);
    hipMemcpyAsync(d_lightMatpinv, _lightMatpinv.data(), M_IMAGE_COUNT * 3 * sizeof(double), hipMemcpyHostToDevice, s1);
    hipMemcpyAsync(d_max, &maxVal, sizeof(double), hipMemcpyHostToDevice, s2);
    hipMemcpyAsync(d_min, &minVal, sizeof(double), hipMemcpyHostToDevice, s3);


    int blocksPerGrid1 = (M_IMAGE_COUNT * _size + threadsPerBlock1 - 1) / threadsPerBlock1;
    dim3 blocksPerGrid((_size + threadsPerBlock2.x - 1) / threadsPerBlock2.x, (M_IMAGE_COUNT + threadsPerBlock2.y - 1) / threadsPerBlock2.y);
    dim3 blocksPerGrid2((_size + threadsPerBlock2.x - 1) / threadsPerBlock2.x, (3 + threadsPerBlock2.y - 1) / threadsPerBlock2.y);
    dim3 blocksPerGrid3((_rows + threadsPerBlock2.x - 1) / threadsPerBlock2.x, (_cols + threadsPerBlock2.y - 1) / threadsPerBlock2.y);
    dim3 blocksPerGrid4((_cols + threadsPerBlock2.x - 1) / threadsPerBlock2.x, (_rows + threadsPerBlock2.y - 1) / threadsPerBlock2.y);

    copyMatrixToCUDA << <blocksPerGrid, threadsPerBlock2 >> > (d_image0Data, d_image1Data, d_image2Data, d_image3Data, d_merged_matrix, _size);
    normalizeImage << <blocksPerGrid1, threadsPerBlock1 >> > (d_merged_matrix, M_IMAGE_COUNT * _size);
    matrixMultiplyKernel << <blocksPerGrid2, threadsPerBlock2 >> > (d_merged_matrix, d_lightMatpinv, d_rho_t, _size, M_IMAGE_COUNT, 3);

    blocksPerGrid1 = (_size + threadsPerBlock1 - 1) / threadsPerBlock1;

    rowNormKernel << <blocksPerGrid1, threadsPerBlock1 >> > (&d_rho_t[2 * _size], d_norm, _size, 1);
    clipKernel << <blocksPerGrid1, threadsPerBlock1 >> > (d_norm, d_norm_t, _size);
    reshapeMatrixKernel << <blocksPerGrid3, threadsPerBlock2 >> > (d_norm_t, d_matrix, _cols, _rows);
    flipKernel << <blocksPerGrid4, threadsPerBlock2 >> > (d_matrix, d_result, _rows, _cols);

    blocksPerGrid1 = (3 + threadsPerBlock1 - 1) / threadsPerBlock1;

    elementWiseDivisionKernel << <blocksPerGrid2, threadsPerBlock2 >> > (d_rho_t, d_norm_t, d_rho, _size, 3);
    setZerosToOnes << <blocksPerGrid1, threadsPerBlock1 >> > (d_rho, _size, 3);
    copyKernel << <blocksPerGrid3, threadsPerBlock2 >> > (d_rho, d_n, _rows, _cols);
    minMaxKernel << <blocksPerGrid3, threadsPerBlock2 >> > (d_n, _rows, _cols, d_max, d_min);
    normalizeKernel << <blocksPerGrid3, threadsPerBlock2 >> > (d_n, d_output, _rows, _cols, d_max, d_min);

    cv::Mat cvMatResult(_rows, _cols, CV_8UC1), _normalmap(_rows, _cols, CV_8UC3);

    hipMemcpyAsync(cvMatResult.data, d_result, _size * sizeof(uchar), hipMemcpyDeviceToHost, s1);
    hipMemcpyAsync(_normalmap.data, d_output, _rows * _cols * sizeof(uchar3), hipMemcpyDeviceToHost, s2);

    /////////////////////////////////////////////////////////////////////////////////////////////




    cv::imwrite("result/0_albedo0.bmp", cvMatResult);
    cv::imwrite("result/0_albedo1.bmp", _normalmap);

    _tend = clock();
    cout << "����ð� : " << (float)(_tend - _tstart) / 1000 << " s" << endl;

    hipFree(d_lightMatpinv);
    hipFree(d_merged_matrix);
    hipFree(d_norm);
    hipFree(d_matrix);
    hipFree(d_result);
    hipFree(d_rho_t);
    hipFree(d_norm_t);
    hipFree(d_rho);
    hipFree(d_n);
    hipFree(d_output);
    hipFree(d_min);
    hipFree(d_max);

    hipStreamDestroy(s1);
    hipStreamDestroy(s2);

    waitKey(1000);

}